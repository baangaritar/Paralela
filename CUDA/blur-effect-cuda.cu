#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
#include <stdio.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>
#include <math.h>
#include <iostream>
#include <pthread.h>
#include <unistd.h>



/* COMPILAR nvcc blur-effect-cuda.cu -o blur-effect-cuda `pkg-config opencv --cflags --libs`*/

__global__ void blurEffect3(uchar *DataIn,uchar *DataOut, int w, int h, int hilos, int bloques  )
{
    //int kernel[9]={2,3,2 };
    int d=7;
    //Fila actual
    int intervalo=w*h/(hilos*bloques);
    int idx = blockIdx.x* blockDim.x + threadIdx.x;
    for (int i = intervalo*idx; i < (intervalo*idx)+intervalo; i+=1) {

      //Indices de los pixeles vecinos

      int idu = i-w;
      int idd = i+w;

      //Valores RGB de cada pixel
      int b = DataIn[3*i]*3;
      int g = DataIn[3*i+1]*3;
      int r = DataIn[3*i+2]*3;


      int bu = DataIn[3*idu]*2;
      int gu = DataIn[3*idu+1]*2;
      int ru = DataIn[3*idu+2]*2;

      int bd = DataIn[3*idd]*2;
      int gd = DataIn[3*idd+1]*2;
      int rd = DataIn[3*idd+2]*2;



      //Promedio de cada valor RGB
      int promR= (r+ru+rd)/d;
      int promG= (g+gu+gd)/d;
      int promB= (b+bu+bd)/d;


      //Asignacion en nueva imagen
      DataOut[3*i] = (unsigned char)(promB);

      DataOut[3*i+1] = (unsigned char)(promG);

      DataOut[3*i+2] = (unsigned char)(promR);

    }


}


__global__ void blurEffect5(uchar *DataIn,uchar *DataOut, int w, int h, int hilos, int bloques  )
{
    //int kernel[9]={ ,2,
      //             2,3,2,
        //             2, };
    int d=11;
    //Fila actual

    int intervalo=w*h/(hilos*bloques);
    int idx = blockIdx.x* blockDim.x + threadIdx.x;
    for (int i = intervalo*idx; i < (intervalo*idx)+intervalo; i+=1) {

      //Indices de los pixeles vecinos
      int idu = i-w;
      int idr = i+3;
      int idl = i-3;
      int idd = i+w;


      //Valores RGB de cada pixel
      int b = DataIn[3*i]*3;
      int g = DataIn[3*i+1]*3;
      int r = DataIn[3*i+2]*3;

      int bu = DataIn[3*idu]*2;
      int gu = DataIn[3*idu+1]*2;
      int ru = DataIn[3*idu+2]*2;

      int bd = DataIn[3*idd]*2;
      int gd = DataIn[3*idd+1]*2;
      int rd = DataIn[3*idd+2]*2;

      int bl = DataIn[3*idl]*2;
      int gl = DataIn[3*idl+1]*2;
      int rl = DataIn[3*idl+2]*2;

      int br = DataIn[3*idr]*2;
      int gr = DataIn[3*idr+1]*2;
      int rr = DataIn[3*idr+2]*2;



      //Promedio de cada valor RGB
      int promR= (r+ru+rd+rl+rr)/d;
      int promG= (g+gu+gd+gl+gr)/d;
      int promB= (b+bu+bd+bl+br)/d;


      //Asignacion en nueva imagen
      DataOut[3*i] = (unsigned char)(promB);

      DataOut[3*i+1] = (unsigned char)(promG);

      DataOut[3*i+2] = (unsigned char)(promR);

    }


}


__global__ void blurEffect7(uchar *DataIn,uchar *DataOut, int w, int h, int hilos, int bloques  )
{
    //int kernel[9]={ ,2,1,
      //             2,3,2,
        //           1,2, };
    int d=13;
    //Fila actual

    int intervalo=w*h/(hilos*bloques);
    int idx = blockIdx.x* blockDim.x + threadIdx.x;
    for (int i = intervalo*idx; i < (intervalo*idx)+intervalo; i+=1) {

      //Indices de los pixeles vecinos
      int idu = i-w;
      int idr = i+3;
      int idl = i-3;
      int idd = i+w;
      int idul = (i-3)-w;
      int iddr = (i+3)+w;

      //Valores RGB de cada pixel
      int b = DataIn[3*i]*3;
      int g = DataIn[3*i+1]*3;
      int r = DataIn[3*i+2]*3;

      int bu = DataIn[3*idu]*2;
      int gu = DataIn[3*idu+1]*2;
      int ru = DataIn[3*idu+2]*2;


      int bul = DataIn[3*idul]*1;
      int gul = DataIn[3*idul+1]*1;
      int rul = DataIn[3*idul+2]*1;

      int bd = DataIn[3*idd]*2;
      int gd = DataIn[3*idd+1]*2;
      int rd = DataIn[3*idd+2]*2;

      int bdr = DataIn[3*iddr]*1;
      int gdr = DataIn[3*iddr+1]*1;
      int rdr = DataIn[3*iddr+2]*1;


      int bl = DataIn[3*idl]*2;
      int gl = DataIn[3*idl+1]*2;
      int rl = DataIn[3*idl+2]*2;

      int br = DataIn[3*idr]*2;
      int gr = DataIn[3*idr+1]*2;
      int rr = DataIn[3*idr+2]*2;



      //Promedio de cada valor RGB
      int promR= (r+ru+rd+rl+rr+rul+rdr)/d;
      int promG= (g+gu+gd+gl+gr+gul+gdr)/d;
      int promB= (b+bu+bd+bl+br+bul+bdr)/d;


      //Asignacion en nueva imagen
      DataOut[3*i] = (unsigned char)(promB);

      DataOut[3*i+1] = (unsigned char)(promG);

      DataOut[3*i+2] = (unsigned char)(promR);

    }


}

__global__ void blurEffect9(uchar *DataIn,uchar *DataOut, int w, int h, int hilos, int bloques  )
{
    //int kernel[9]={1,2,1,
      //             2,3,2,
        //           1,2,1};
    int d=15;
    //Fila actual

    int intervalo=w*h/(hilos*bloques);
    int idx = blockIdx.x* blockDim.x + threadIdx.x;
    for (int i = intervalo*idx; i < (intervalo*idx)+intervalo; i+=1) {

      //Indices de los pixeles vecinos
      int idu = i-w;
      int idr = i+3;
      int idl = i-3;
      int idd = i+w;

      int idul = (i-3)-w;
      int idur = (i+3)-w;
      int iddl = (i-3)+w;
      int iddr = (i+3)+w;

      //Valores RGB de cada pixel
      int b = DataIn[3*i]*3;
      int g = DataIn[3*i+1]*3;
      int r = DataIn[3*i+2]*3;

      int bu = DataIn[3*idu]*2;
      int gu = DataIn[3*idu+1]*2;
      int ru = DataIn[3*idu+2]*2;


      int bul = DataIn[3*idul]*1;
      int gul = DataIn[3*idul+1]*1;
      int rul = DataIn[3*idul+2]*1;


      int bur = DataIn[3*idur]*1;
      int gur = DataIn[3*idur+1]*1;
      int rur = DataIn[3*idur+2]*1;


      int bd = DataIn[3*idd]*2;
      int gd = DataIn[3*idd+1]*2;
      int rd = DataIn[3*idd+2]*2;


      int bdl = DataIn[3*iddl]*1;
      int gdl = DataIn[3*iddl+1]*1;
      int rdl = DataIn[3*iddl+2]*1;

      int bdr = DataIn[3*iddr]*1;
      int gdr = DataIn[3*iddr+1]*1;
      int rdr = DataIn[3*iddr+2]*1;


      int bl = DataIn[3*idl]*2;
      int gl = DataIn[3*idl+1]*2;
      int rl = DataIn[3*idl+2]*2;

      int br = DataIn[3*idr]*2;
      int gr = DataIn[3*idr+1]*2;
      int rr = DataIn[3*idr+2]*2;



      //Promedio de cada valor RGB
      int promR= (r+ru+rd+rl+rr+rur+rul+rdl+rdr)/d;
      int promG= (g+gu+gd+gl+gr+gur+gul+gdl+gdr)/d;
      int promB= (b+bu+bd+bl+br+bur+bul+bdl+bdr)/d;


      //Asignacion en nueva imagen
      DataOut[3*i] = (unsigned char)(promB);

      DataOut[3*i+1] = (unsigned char)(promG);

      DataOut[3*i+2] = (unsigned char)(promR);

    }


}

__global__ void blurEffect11(unsigned char *DataIn, unsigned char *DataOut, int w, int h, int hilos, int bloques)
{
    //int kernel[9]={1,2,1,
      //             2,3,2,1
        //           1,2,1
                   //  1};

    int d=17;

    int intervalo=w*h/(hilos*bloques);
    int idx = blockIdx.x* blockDim.x + threadIdx.x;
    for (int i = intervalo*idx; i < (intervalo*idx)+intervalo; i+=1){

      //Piexeles alrededor
      int idu = i-w;
      int idr = i+3;
      int idl = i-3;
      int idd = i+w;

      int idul = (i-3)-w;
      int idur = (i+3)-w;
      int iddl = (i-3)+w;
      int iddr = (i+3)+w;


      int idd2 = i+w+w;
      int idr2 = i+6;


      //obtencion RGB de cada pixel y multiplicacion por el valor del kernel gaussiano

      int b = DataIn[3*i]*3;
      int g = DataIn[3*i+1]*3;
      int r = DataIn[3*i+2]*3;

      int bu = DataIn[3*idu]*2;
      int gu = DataIn[3*idu+1]*2;
      int ru = DataIn[3*idu+2]*2;


      int bul = DataIn[3*idul]*1;
      int gul = DataIn[3*idul+1]*1;
      int rul = DataIn[3*idul+2]*1;


      int bur = DataIn[3*idur]*1;
      int gur = DataIn[3*idur+1]*1;
      int rur = DataIn[3*idur+2]*1;


      int bd = DataIn[3*idd]*2;
      int gd = DataIn[3*idd+1]*2;
      int rd = DataIn[3*idd+2]*2;


      int bdl = DataIn[3*iddl]*1;
      int gdl = DataIn[3*iddl+1]*1;
      int rdl = DataIn[3*iddl+2]*1;

      int bdr = DataIn[3*iddr]*1;
      int gdr = DataIn[3*iddr+1]*1;
      int rdr = DataIn[3*iddr+2]*1;


      int bl = DataIn[3*idl]*2;
      int gl = DataIn[3*idl+1]*2;
      int rl = DataIn[3*idl+2]*2;

      int br = DataIn[3*idr]*2;
      int gr = DataIn[3*idr+1]*2;
      int rr = DataIn[3*idr+2]*2;


      int br2=DataIn[3*idr2];
      int gr2=DataIn[3*idr2+1];
      int rr2=DataIn[3*idr2+2];


      int bd2=DataIn[3*idd2];
      int gd2=DataIn[3*idd2+1];
      int rd2=DataIn[3*idd2+2];




      //Promedio de cada valor R G y B de cada pixel

      int promR= (r+ru+rd+rl+rr+rur+rul+rdl+rdr+rr2+rd2)/d;
      int promG= (g+gu+gd+gl+gr+gur+gul+gdl+gdr+gr2+gd2)/d;
      int promB= (b+bu+bd+bl+br+bur+bul+bdl+bdr+br2+bd2)/d;

      //Asignacion en la salida

      DataOut[3*i] = (unsigned char)(promB);

      DataOut[3*i+1] = (unsigned char)(promG);

      DataOut[3*i+2] = (unsigned char)(promR);
    }

}


__global__ void blurEffect13(unsigned char *DataIn, unsigned char *DataOut, int w, int h, int hilos, int bloques)
{
    //int kernel[9]={1,2,1,1
      //             2,3,2,1
        //           1,2,1
      //             1 1    };

    int d=19;

    int intervalo=w*h/(hilos*bloques);
    int idx = blockIdx.x* blockDim.x + threadIdx.x;
    for (int i = intervalo*idx; i < (intervalo*idx)+intervalo; i+=1){

      //Piexeles alrededor
      int idu = i-w;
      int idr = i+3;
      int idl = i-3;
      int idd = i+w;

      int idul = (i-3)-w;
      int idur = (i+3)-w;
      int iddl = (i-3)+w;
      int iddr = (i+3)+w;


      int idd2 = i+w+w;
      int iddl2 = (i-3)+w+w;

      int idur2 = (i+6)-w;
      int idr2 = i+6;

      //obtencion RGB de cada pixel y multiplicacion por el valor del kernel gaussiano

      int b = DataIn[3*i]*3;
      int g = DataIn[3*i+1]*3;
      int r = DataIn[3*i+2]*3;

      int bu = DataIn[3*idu]*2;
      int gu = DataIn[3*idu+1]*2;
      int ru = DataIn[3*idu+2]*2;


      int bul = DataIn[3*idul]*1;
      int gul = DataIn[3*idul+1]*1;
      int rul = DataIn[3*idul+2]*1;


      int bur = DataIn[3*idur]*1;
      int gur = DataIn[3*idur+1]*1;
      int rur = DataIn[3*idur+2]*1;


      int bd = DataIn[3*idd]*2;
      int gd = DataIn[3*idd+1]*2;
      int rd = DataIn[3*idd+2]*2;


      int bdl = DataIn[3*iddl]*1;
      int gdl = DataIn[3*iddl+1]*1;
      int rdl = DataIn[3*iddl+2]*1;

      int bdr = DataIn[3*iddr]*1;
      int gdr = DataIn[3*iddr+1]*1;
      int rdr = DataIn[3*iddr+2]*1;


      int bl = DataIn[3*idl]*2;
      int gl = DataIn[3*idl+1]*2;
      int rl = DataIn[3*idl+2]*2;

      int br = DataIn[3*idr]*2;
      int gr = DataIn[3*idr+1]*2;
      int rr = DataIn[3*idr+2]*2;



      int bur2=DataIn[3*idur2];
      int gur2=DataIn[3*idur2+1];
      int rur2=DataIn[3*idur2+2];


      int br2=DataIn[3*idr2];
      int gr2=DataIn[3*idr2+1];
      int rr2=DataIn[3*idr2+2];

      int bdl2=DataIn[3*iddl2];
      int gdl2=DataIn[3*iddl2+1];
      int rdl2=DataIn[3*iddl2+2];

      int bd2=DataIn[3*idd2];
      int gd2=DataIn[3*idd2+1];
      int rd2=DataIn[3*idd2+2];





      //Promedio de cada valor R G y B de cada pixel

      int promR= (r+ru+rd+rl+rr+rur+rul+rdl+rdr+rur2+rr2+rdl2+rd2)/d;
      int promG= (g+gu+gd+gl+gr+gur+gul+gdl+gdr+gur2+gr2+gdl2+gd2)/d;
      int promB= (b+bu+bd+bl+br+bur+bul+bdl+bdr+bur2+br2+bdl2+bd2)/d;

      //Asignacion en la salida

      DataOut[3*i] = (unsigned char)(promB);

      DataOut[3*i+1] = (unsigned char)(promG);

      DataOut[3*i+2] = (unsigned char)(promR);
    }

}


__global__ void blurEffect15(unsigned char *DataIn, unsigned char *DataOut, int w, int h, int hilos, int bloques)
{
    //int kernel[9]={1,2,1,
      //             2,3,2,
        //           1,2,1};

    int d=21;

    int intervalo=w*h/(hilos*bloques);
    int idx = blockIdx.x* blockDim.x + threadIdx.x;
    for (int i = intervalo*idx; i < (intervalo*idx)+intervalo; i+=1){

      //Piexeles alrededor
      int idu = i-w;
      int idr = i+3;
      int idl = i-3;
      int idd = i+w;

      int idul = (i-3)-w;
      int idur = (i+3)-w;
      int iddl = (i-3)+w;
      int iddr = (i+3)+w;


      int idd2 = i+w+w;
      int iddl2 = (i-3)+w+w;
      int iddr2 = (i+3)+w+w;

      int idur2 = (i+6)-w;
      int idr2 = i+6;
      int iddrd2 = (i+3)+w+w;

      //obtencion RGB de cada pixel y multiplicacion por el valor del kernel gaussiano

      int b = DataIn[3*i]*3;
      int g = DataIn[3*i+1]*3;
      int r = DataIn[3*i+2]*3;

      int bu = DataIn[3*idu]*2;
      int gu = DataIn[3*idu+1]*2;
      int ru = DataIn[3*idu+2]*2;


      int bul = DataIn[3*idul]*1;
      int gul = DataIn[3*idul+1]*1;
      int rul = DataIn[3*idul+2]*1;


      int bur = DataIn[3*idur]*1;
      int gur = DataIn[3*idur+1]*1;
      int rur = DataIn[3*idur+2]*1;


      int bd = DataIn[3*idd]*2;
      int gd = DataIn[3*idd+1]*2;
      int rd = DataIn[3*idd+2]*2;


      int bdl = DataIn[3*iddl]*1;
      int gdl = DataIn[3*iddl+1]*1;
      int rdl = DataIn[3*iddl+2]*1;

      int bdr = DataIn[3*iddr]*1;
      int gdr = DataIn[3*iddr+1]*1;
      int rdr = DataIn[3*iddr+2]*1;


      int bl = DataIn[3*idl]*2;
      int gl = DataIn[3*idl+1]*2;
      int rl = DataIn[3*idl+2]*2;

      int br = DataIn[3*idr]*2;
      int gr = DataIn[3*idr+1]*2;
      int rr = DataIn[3*idr+2]*2;



      int bur2=DataIn[3*idur2];
      int gur2=DataIn[3*idur2+1];
      int rur2=DataIn[3*idur2+2];


      int br2=DataIn[3*idr2];
      int gr2=DataIn[3*idr2+1];
      int rr2=DataIn[3*idr2+2];

      int bdr2=DataIn[3*iddr2];
      int gdr2=DataIn[3*iddr2+1];
      int rdr2=DataIn[3*iddr2+2];


      int bdl2=DataIn[3*iddl2];
      int gdl2=DataIn[3*iddl2+1];
      int rdl2=DataIn[3*iddl2+2];

      int bd2=DataIn[3*idd2];
      int gd2=DataIn[3*idd2+1];
      int rd2=DataIn[3*idd2+2];



      int bdrd2=DataIn[3*iddrd2];
      int gdrd2=DataIn[3*iddrd2+1];
      int rdrd2=DataIn[3*iddrd2+2];




      //Promedio de cada valor R G y B de cada pixel

      int promR= (r+ru+rd+rl+rr+rur+rul+rdl+rdr+rur2+rr2+rdr2+rdl2+rd2+rdrd2)/d;
      int promG= (g+gu+gd+gl+gr+gur+gul+gdl+gdr+gur2+gr2+gdr2+gdl2+gd2+gdrd2)/d;
      int promB= (b+bu+bd+bl+br+bur+bul+bdl+bdr+bur2+br2+bdr2+bdl2+bd2+bdrd2)/d;

      //Asignacion en la salida

      DataOut[3*i] = (unsigned char)(promB);

      DataOut[3*i+1] = (unsigned char)(promG);

      DataOut[3*i+2] = (unsigned char)(promR);
    }

}





int main(int argc, char** argv)
{
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);

        int height,width;

        char *imgIN;
        imgIN = new char [1024];

        imgIN=argv[1];
        const char* filename1 = argv[2];
        int kernelSize=atoi(argv[3]);
        int NumThreadsX=atoi(argv[4]);

        //Imagen de entrada
        IplImage* image;
        image = cvLoadImage(imgIN, 1);

        height = image->height;
        width = image->width;

        int step = image->widthStep;
        int SizeIn = (step*height);
        printf("\nProcessing image\n");



        //imagen de salida

        IplImage *image2 = cvCreateImage(cvSize(width, height), IPL_DEPTH_8U, 3);
        int step2 = image2->widthStep;
        int SizeOut = step2 * height;


        //GPU
        uchar4* DatIn = (uchar4*)image->imageData;
        unsigned char * DatOut = (unsigned char*)image2->imageData;
        unsigned char *datIndev;
        unsigned char *datOutdev;

        printf("Allocating memory on Device\n");

        // Reservar memoria en GPU


        hipMalloc(&datIndev, SizeIn * sizeof(unsigned char));
        hipMalloc(&datOutdev, SizeOut * sizeof(unsigned char));


        printf("Copy data on Device\n");


        // Copiar datos en GPU

        hipMemcpy(datIndev, DatIn, SizeIn * sizeof(unsigned char), hipMemcpyHostToDevice);
        hipMemcpy(datOutdev, DatOut, SizeOut * sizeof(unsigned char), hipMemcpyHostToDevice);


        //kernel CUDA

        //int NumThreadsX = deviceProp.maxThreadsPerBlock;
        int NumBlocksX =50;
        dim3 blocks(NumBlocksX);
        dim3 threads(NumThreadsX);


        switch(kernelSize){
            case 3:
              blurEffect3<<<blocks, threads>>>(datIndev, datOutdev, width,height,NumThreadsX,NumBlocksX);
              break;
            case 5:
              blurEffect5<<<blocks, threads>>>(datIndev, datOutdev, width,height,NumThreadsX,NumBlocksX);
              break;
            case 7:
              blurEffect7<<<blocks, threads>>>(datIndev, datOutdev, width,height,NumThreadsX,NumBlocksX);
              break;
            case 9:
              blurEffect9<<<blocks, threads>>>(datIndev, datOutdev, width,height,NumThreadsX,NumBlocksX);
              break;
            case 11:
              blurEffect11<<<blocks, threads>>>(datIndev, datOutdev, width,height,NumThreadsX,NumBlocksX);
              break;
            case 13:
              blurEffect13<<<blocks, threads>>>(datIndev, datOutdev, width,height,NumThreadsX,NumBlocksX);
              break;
            case 15:
              blurEffect15<<<blocks, threads>>>(datIndev, datOutdev, width,height,NumThreadsX,NumBlocksX);
              break;
        }



        //Obtener resultado de GPU a HOST

        hipMemcpy(DatOut, datOutdev, SizeOut * sizeof(unsigned char), hipMemcpyDeviceToHost);

        //Mostrar iamgen
        cvNamedWindow("Image original");
        cvShowImage("Image original", image);
        cvNamedWindow("Image blur");
        cvShowImage("Image blur", image2);

        //Guardar imagen


        cvSaveImage( filename1, image2 );

        //Liberar memoria en GPU
        hipFree(datOutdev);
        hipFree(datIndev);
        cvWaitKey(0);
        return 0;
}
